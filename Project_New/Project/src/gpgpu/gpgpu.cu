#include "hip/hip_runtime.h"
#include <gpgpu.h>
#include <algorithm>
#include <iostream>
#include <random>

//float4 FOX_COLOR = make_float4(1.0f, 0.5f, 0.0f, 1.0f);



// Override - 
__device__ float2 operator-(float2 a, float2 b) {
	return make_float2(a.x - b.x, a.y - b.y);
};

void GetGPGPUInfo() {
	hipDeviceProp_t cuda_propeties;
	hipGetDeviceProperties(&cuda_propeties, 0);
	std::cout << "maxThreadsPerBlock: " << cuda_propeties.maxThreadsPerBlock << std::endl;
}

__global__ void kernel_uv(hipSurfaceObject_t surface, int32_t width, int32_t height, float time) {
	int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	float u = (float)x / width;
	float v = (float)y / height;
	float4 color = make_float4(u, v, cos(time), 1.0f);
	surf2Dwrite(color, surface, x * sizeof(float4), y);
}

__global__ void kernel_copy(hipSurfaceObject_t surface_in, hipSurfaceObject_t surface_out) {
	int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	float4 color = make_float4(1.f, 0.f, 1.f, 1.0f);
	surf2Dread(&color, surface_in, x * sizeof(float4), y);
	surf2Dwrite(color, surface_out, x * sizeof(float4), y);
}


// kernel of draw map
__global__ void kernel_draw_map(hipSurfaceObject_t surface) {
	int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	float4 color = make_float4(0.6f, 0.9f, 0.05f, 1.0f);

	surf2Dwrite(color, surface, x * sizeof(float4), y);
}

// draw a circle 
/*
__global__ void drawCircleKernel(hipSurfaceObject_t surface, int centerX, int centerY, int radius, float4 color) {
	// calculate the x and y coordinates for the current thread
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	Fox myfox = { 120.0, 120.0, 10 };
	

	// check if the current pixel is within the circle
	//if (hypotf(centerX - x, centerY - y) < radius) {
		//surf2Dwrite(color, surface, sizeof(float4) * x, y, hipBoundaryModeTrap);
	//}

	if (hypotf(myfox.u - x, myfox.v - y) < myfox.radius) {
		surf2Dwrite(color, surface, sizeof(float4) * x, y, hipBoundaryModeTrap);
	}

}
*/

// define the kernel
__global__ void drawCircleKernel(hipSurfaceObject_t surface, Fox* fox_buffer, Rabbit* rabbit_buffer, float4 fox_color, float4 rabbit_color, int number_foxes, int number_rabbits) {
	// calculate the x and y coordinates for the current thread
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	// iterate through the buffer
	for (int i = 0; i < MAX_FOX; i++) {

		if (fox_buffer[i].is_alive == true){
			if (hypotf(fox_buffer[i].u - x, fox_buffer[i].v - y) < fox_buffer[i].radius) {
				surf2Dwrite(fox_color, surface, sizeof(float4) * x, y, hipBoundaryModeTrap);
			}
			// Changing positions of fox
			if (fox_buffer[i].u >= 1000 || fox_buffer[i].u <= 10){
				fox_buffer[i].direction_u = 0.0f - fox_buffer[i].direction_u;
			}
			// Change of direction
			if (fox_buffer[i].v >= 1000 || fox_buffer[i].v <= 10){
				fox_buffer[i].direction_v = 0.0f - fox_buffer[i].direction_v;
			} 
			fox_buffer[i].u = fox_buffer[i].u + fox_buffer[i].direction_u;
			fox_buffer[i].v = fox_buffer[i].v + fox_buffer[i].direction_v;
		}

	}

	for (int i = 0; i < MAX_RABBIT; i++) {
		if (rabbit_buffer[i].is_alive == true){
			if ((hypotf(rabbit_buffer[i].u - x, rabbit_buffer[i].v - y) < rabbit_buffer[i].radius)) {
				surf2Dwrite(rabbit_color, surface, sizeof(float4) * x, y, hipBoundaryModeTrap);
			}

			// Changing positions of rabbit
			if (rabbit_buffer[i].u >= 1000 || rabbit_buffer[i].u <= 10){
				rabbit_buffer[i].direction_u = 0.0f - rabbit_buffer[i].direction_u;
			}
			
			if (rabbit_buffer[i].v >= 1000 || rabbit_buffer[i].v <= 10){
				rabbit_buffer[i].direction_v = 0.0f - rabbit_buffer[i].direction_v;
			} 
			// Changing positions of rabbit
			rabbit_buffer[i].u = rabbit_buffer[i].u + rabbit_buffer[i].direction_u;
			rabbit_buffer[i].v = rabbit_buffer[i].v + rabbit_buffer[i].direction_v;
			}

	}


}
  



void DrawUVs(hipSurfaceObject_t surface, int32_t width, int32_t height, float time) {
	dim3 threads(32, 32);
	dim3 blocks(32, 32);
	kernel_uv << <blocks, threads >> > (surface, width, height, time);
}

void CopyTo(hipSurfaceObject_t surface_in, hipSurfaceObject_t surface_out, int32_t width, int32_t height) {
	dim3 threads(32, 32);
	dim3 blocks(32, 32);
	kernel_copy << <blocks, threads >>> (surface_in, surface_out);
}
/*
// Init foxes 
__global__ void kernel_initFoxes(Fox* foxes, int numFoxes)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numFoxes)
    {
        std::mt19937_64 gen(clock());
        std::uniform_real_distribution<float> uDist(0.0f, 1024);
        std::uniform_real_distribution<float> vDist(0.0f, 1024);
        std::uniform_real_distribution<float> rDist(0.0f, 50);

        foxes[idx].u = uDist(gen);
        foxes[idx].v = vDist(gen);
        foxes[idx].radius = rDist(gen);
    }
}*/

/* Working one
// Added a draw map 
void DrawMap(hipSurfaceObject_t surface, int32_t width, int32_t height, float time) {

	dim3 threads(32, 32);
	dim3 blocks(32, 32);

	kernel_draw_map << <blocks, threads >> > (surface);

	// Init foxes
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_real_distribution<> x(1.0, 500.0);
	std::uniform_real_distribution<> y(1.0, 500.0);
	Fox* fox_buffer = new Fox[50];
	for (int i = 0; i < 50; i++) {
		fox_buffer[i].u = x(gen);
		fox_buffer[i].v = y(gen);
		fox_buffer[i].radius = 50;
	}




	drawCircleKernel << <blocks, threads >> > (surface, 150, 512, 50, make_float4(1.0f, 0.5f, 0.0f, 1.0f));
	// using buffer
	//drawCircleKernel << <blocks, threads >> > (surface, fox_buffer, make_float4(1.0f, 0.5f, 0.0f, 1.0f));

}*/
/*
void init(Fox* fox_buffer, Rabbit* rabbit_buffer, Fox** device_foxes, Rabbit** device_rabbits) {

	int number_foxes = 6;
	int number_rabbits = 50;
	// Init foxes
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_real_distribution<> x(0.0, 1023.0);
	std::uniform_real_distribution<> y(0.0, 1023.0);
	// host foxes
	
	for (int i = 0; i < number_foxes; i++) {
		fox_buffer[i].u = x(gen);
		fox_buffer[i].v = y(gen);
		fox_buffer[i].radius = 15;
	}

	fox_buffer[0].u = 100;
	fox_buffer[0].v = 100;


	for (int i = 0; i < number_rabbits; i++) {
		rabbit_buffer[i].u = x(gen);
		rabbit_buffer[i].v = y(gen);
		rabbit_buffer[i].radius = 15;
	}

	// Allocate device-side memory

	hipMalloc(device_foxes, sizeof(Fox) * number_foxes);

	
	hipMalloc(device_rabbits, sizeof(Rabbit) * number_rabbits);

	// copy data to device
	hipMemcpy(device_foxes, fox_buffer, sizeof(Fox) * number_foxes, hipMemcpyHostToDevice);
	hipMemcpy(device_rabbits, rabbit_buffer, sizeof(Rabbit) * number_rabbits, hipMemcpyHostToDevice);

	// copy data back to host
	//hipMemcpy(fox_buffer, device_foxes, sizeof(Fox) * number_foxes, hipMemcpyDeviceToHost);
	//hipMemcpy(rabbit_buffer, device_rabbits, sizeof(Rabbit) * number_rabbits, hipMemcpyDeviceToHost);

	
}
*/
void destroy(Fox* device_foxes, Rabbit* device_rabbits) {
	// free device-side memory
	hipFree(device_foxes);
	hipFree(device_rabbits);
}



//Trying to make the buffer work
void DrawMap(hipSurfaceObject_t surface, int32_t width, int32_t height, float time, int number_foxes, int number_rabbits) {

	dim3 threads(32, 32);
	dim3 blocks(32, 32);


	float4 fox_color = make_float4(1.0f, 0.5f, 0.0f, 1.0f);
	float4 rabbit_color = make_float4(1.0f, 1.0f, 1.0f, 1.0f);

	kernel_draw_map << <blocks, threads >> > (surface);
	
	static bool is_init = false;
	static Fox* device_foxes;
	static Rabbit* device_rabbits;
	//init only once then jump always
	if (!is_init) {
		is_init = true;
		// Init foxes
		std::random_device rd;
		std::mt19937 gen(rd());
		std::uniform_real_distribution<> x(1.0, 1024.0);
		std::uniform_real_distribution<> y(1.0, 1024.0);
		// direction u v
		std::uniform_real_distribution<> direction_x(-0.001, 0.001);
		std::uniform_real_distribution<> direction_y(-0.001, 0.001);
		// host foxes
		Fox* fox_buffer = new Fox[MAX_FOX];
		for (int i = 0; i < MAX_FOX; i++) {
			fox_buffer[i].u = x(gen);
			fox_buffer[i].v = y(gen);
			fox_buffer[i].direction_u = direction_x(gen);
			fox_buffer[i].direction_v = direction_y(gen);
			fox_buffer[i].radius = 15;
			if (i < number_foxes){
				fox_buffer[i].is_alive = true;
			}
		}

		// host rabbits
		Rabbit* rabbit_buffer = new Rabbit[MAX_RABBIT];
		for (int i = 0; i < MAX_RABBIT; i++) {
			rabbit_buffer[i].u = x(gen);
			rabbit_buffer[i].v = y(gen);
			rabbit_buffer[i].direction_u = direction_x(gen);
			rabbit_buffer[i].direction_v = direction_y(gen);
			rabbit_buffer[i].radius = 15;
			if (i < number_rabbits){
				rabbit_buffer[i].is_alive = true;
			}
		}

		// Allocate device-side memory
		
		hipMalloc(&device_foxes, sizeof(Fox) * MAX_FOX);

		hipMalloc(&device_rabbits, sizeof(Rabbit) * MAX_RABBIT);

		// copy data to device
		hipMemcpy(device_foxes, fox_buffer, sizeof(Fox) * MAX_FOX, hipMemcpyHostToDevice);
		hipMemcpy(device_rabbits, rabbit_buffer, sizeof(Rabbit) * MAX_RABBIT, hipMemcpyHostToDevice);
	}

	drawCircleKernel << <blocks, threads >> > (surface, device_foxes, device_rabbits, fox_color, rabbit_color, number_foxes, number_rabbits);

	// copy data back to host
	/*hipMemcpy(fox_buffer, device_foxes, sizeof(Fox) * number_foxes, hipMemcpyDeviceToHost);
	hipMemcpy(rabbit_buffer, device_rabbits, sizeof(Rabbit) * number_rabbits, hipMemcpyDeviceToHost);

	// free device-side memory
	hipFree(device_foxes);
	hipFree(device_rabbits);*/
	
}


/*
void DrawAnimal(hipSurfaceObject_t surface, int32_t width, int32_t height, float time) {
	// buffer of Foxes & rabbits
	Fox* fox_buffer = new Fox[6];
	Rabbit* rabbit_buffer = new Rabbit[40];
	float4 rabbit_color = make_float4(0f, 0f, 0f, 1f);
	float4 fox_color = make_float4(255f, 255f, 255f, 1f);

	for (int i = 0; i < 6; i++) {
		std::uniform_real_distribution<> x(1, 1024);
		std::uniform_real_distribution<> y(1, 1024);
		fox_buffer[i].u = x;
		fox_buffer[i].v = y;
		
		surf2Dwrite(fox_color, surface, x * sizeof(float4), y);
	}

	for (int j = 0; j < 40; j++) {
		std::uniform_real_distribution<> x(1, 1024);
		std::uniform_real_distribution<> y(1, 1024);
		rabbit_buffer[i].u = x;
		rabbit_buffer[i].v = y;
		surf2Dwrite(rabbit_color, surface, x * sizeof(float4), y);
	}

	DrawMap(hipSurfaceObject_t surface, int32_t width, int32_t height, float time);
}
*/

