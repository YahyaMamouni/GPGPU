#include <gpgpu.h>
#include <algorithm>
#include <iostream>

void GetGPGPUInfo() {
	hipDeviceProp_t cuda_properties;
	hipGetDeviceProperties(&cuda_properties, 0);
	std::cout << cuda_properties::cudamaxThreadsPerBlock;
}