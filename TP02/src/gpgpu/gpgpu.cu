#include "hip/hip_runtime.h"
#include <gpgpu.h>
#include <algorithm>
#include <iostream>



//Kernel gray-scale

__global__ void kernel_gray(float * device_image_float){
    int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    device_image_float[index] = device_image_float[index] + 0.5f;
}
/*
// Kernel uv
__global__ void kernel_uv(float * image, int32_t width, int32_t height){
    //int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    //int32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    int index;
    for (int j = 0; j < height; j++){
        for (int i = 0 ; i < width ; ++i){
                index = 3 * (width * j + i);
                float u = (float)i / (float)width;
                float v = (float)j / (float)height;
                // (1-u) and (1-v) is added to get the right values because if we keep u and v we will get the image but inverted (Ask teacher)
                // Is it a thing in the gpu ? because u & v worked fine in TP1
                int ir = int(255.0 * (1-u));
                int ig = int(255.0 * (1-v));

                image[index++] = ir;
                image[index++] = ig;
                image[index++] = 0;
        }
    }

}*/

__global__ void kernel_uv(float * image, int32_t width, int32_t height){
    int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    int index;
    index = 3 * (width * y + x);
    float u = (float)x / (float)width;
    float v = (float)y / (float)height;
    int ir = int(255.0 * (1-u));
    int ig = int(255.0 * (1-v));

    image[index++] = ir;
    image[index++] = ig;
    image[index++] = 0;
        
    

}


void GetGPGPUInfo(){
    hipDeviceProp_t cuda_properties;
    hipGetDeviceProperties(&cuda_properties, 0);
    // Display the maxThreadsPerBlock property
    std::cout << "Max threads per block : \n" << cuda_properties.maxThreadsPerBlock;
}

void GenerateGrayscaleImage(std::vector<uint8_t>& host_image_uint8, int32_t width, int32_t height){
    std::vector<float> host_image_float;
    // Filling host_image_float with the content of host_image_uint8
    // Using std::transform
    // the y argument will be the corresponding value in the output image.
    for (int i = 0; i < host_image_uint8.size(); i++)
	{
		float f = host_image_uint8.at(i);
		host_image_float.push_back(f / 255);
	}

    // Asking CUDA to duplicate the new image on the GPU

    // Init a temporary image that we will use
    float * device_image_float = nullptr;
    hipMalloc(&device_image_float, host_image_uint8.size()*sizeof(float));
    // Copy data of host_image_float into device_image_float
    hipMemcpy(
        device_image_float,
        host_image_float.data(),
        host_image_uint8.size() * sizeof(float),
        hipMemcpyHostToDevice);
    // At this point host_image_uint8 and host_image_float are only editable by the host and device_image_float only by the device

    // Call to the kernel gray
    //kernel_gray<<<height*3, 1024>>>(device_image_float);

    // Call the kernel uv
    dim3 threads(32,32);
    dim3 blocks(32, 32);
    kernel_uv <<<blocks, threads>>>(device_image_float, width, height);

    hipMemcpy(
        host_image_float.data(),
        device_image_float,
        host_image_uint8.size() * sizeof(float),
        hipMemcpyDeviceToHost);

    for (int i = 0; i < host_image_float.size(); i++)
	{
		host_image_uint8.at(i) = host_image_float.at(i) * 255;
	}

    //Free the GPU buffer
    hipFree(device_image_float);
}